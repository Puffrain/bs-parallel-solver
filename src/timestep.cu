#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>          /* 为 exp() 提供声明 */
#include <mpi.h>
#include "black_scholes.h"

/*---------------------------------------------*/
/*  串行三对角 Thomas 求解器                    */
/*---------------------------------------------*/
static void thomas_serial(int N, double *a, double *b,
                          double *c, double *d, double *x)
{
    /* 前向消元 */
    for (int i = 1; i < N; ++i) {
        double m = a[i] / b[i - 1];
        b[i]    -= m * c[i - 1];
        d[i]    -= m * d[i - 1];
    }
    /* 回代 */
    x[N - 1] = d[N - 1] / b[N - 1];
    for (int i = N - 2; i >= 0; --i)
        x[i] = (d[i] - c[i] * x[i + 1]) / b[i];
}

/*---------------------------------------------*/
/*  “简单” 并行求解：Gather → Thomas → Scatter  */
/*---------------------------------------------*/
static void solve_tridiag_MPI(const BSGrid *g, double *rhs)
{
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    /* 收集每个进程的行数 */
    int n    = g->nloc;
    int *cnt = NULL, *dis = NULL;
    if (rank == 0) {
        cnt = malloc(size * sizeof(int));
        dis = malloc(size * sizeof(int));
    }
    MPI_Gather(&n, 1, MPI_INT, cnt, 1, MPI_INT, 0, MPI_COMM_WORLD);

    int Ntot = 0;
    if (rank == 0) {
        dis[0] = 0;
        for (int i = 0; i < size; ++i) {
            Ntot += cnt[i];
            if (i > 0) dis[i] = dis[i - 1] + cnt[i - 1];
        }
    }

    /* Gather 本地 a,b,c,rhs 到 rank 0 */
    double *aAll = NULL, *bAll = NULL, *cAll = NULL;
    double *dAll = NULL, *xAll = NULL;

    if (rank == 0) {
        aAll = malloc(Ntot * sizeof(double));
        bAll = malloc(Ntot * sizeof(double));
        cAll = malloc(Ntot * sizeof(double));
        dAll = malloc(Ntot * sizeof(double));
        xAll = malloc(Ntot * sizeof(double));
    }

    MPI_Gatherv(&a_loc[1], n, MPI_DOUBLE, aAll, cnt, dis,
                MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Gatherv(&b_loc[1], n, MPI_DOUBLE, bAll, cnt, dis,
                MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Gatherv(&c_loc[1], n, MPI_DOUBLE, cAll, cnt, dis,
                MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Gatherv(&rhs[1],   n, MPI_DOUBLE, dAll, cnt, dis,
                MPI_DOUBLE, 0, MPI_COMM_WORLD);

    /* rank 0 串行求解 */
    if (rank == 0)
        thomas_serial(Ntot, aAll, bAll, cAll, dAll, xAll);

    /* Scatter 解回各进程 */
    MPI_Scatterv(xAll, cnt, dis, MPI_DOUBLE,
                 &rhs[1], n, MPI_DOUBLE, 0, MPI_COMM_WORLD);

    /* 释放临时缓冲 */
    if (rank == 0) {
        free(aAll); free(bAll); free(cAll);
        free(dAll); free(xAll);
        free(cnt);  free(dis);
    }
}

/* 欧式看涨 Payoff */
static inline double payoff(double S, double K)
{ return (S > K) ? (S - K) : 0.0; }

/*---------------------------------------------*/
/*           隐式欧拉时间推进主函数             */
/*---------------------------------------------*/
void time_stepper(const BSParams *p, const BSGrid *g, BSTime *t)
{
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int n = g->nloc;          /* 本进程实节点数 */
    int N = n + 2;            /* 含 2 个 ghost */

    t->V_old = calloc(N, sizeof(double));
    if (!t->V_old) { perror("malloc"); MPI_Abort(MPI_COMM_WORLD, 1); }

    /* t=T Payoff */
    for (int i = 0; i < n; ++i)
        t->V_old[i + 1] = payoff(g->S[i + 1], p->K);

    double dt = p->T / p->N_t;

    for (int step = 1; step <= p->N_t; ++step)
    {
        double t_remain = p->T - step * dt;

        /* 左边界 */
        if (rank == 0) t->V_old[1] = 0.0;

        /* 右边界 */
        if (rank == size - 1) {
            double Smax = g->S[n];
            t->V_old[n] = Smax - p->K * exp(-p->r * t_remain);
        }

        /* 求解 A·V_new = V_old (结果覆盖到 V_old[1..n]) */
        solve_tridiag_MPI(g, t->V_old);

        if (rank == 0 && (step % 100 == 0 || step == p->N_t))
            printf("隐式步 %d / %d 完成\n", step, p->N_t);
    }

    /* 使 V_new 指向解向量首元素，便于主程序 free */
    t->V_new = &t->V_old[1];
    if (rank == 0) puts("隐式欧拉并行求解结束 ✅");
}

